#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#ifdef HAVE_CONFIG_H
#include "config.h"
#endif

#include "iordef.h"

#define RANDALGO_GOLDEN_RATIO_PRIME        0x9e37fffffffc0001UL

__global__ 
void cu_generate_memory_timestamp(float * buf, size_t bytes, int rand_seed, int pretendRank){
  
}

extern "C" void generate_memory_pattern_gpu(char * buf, size_t bytes, int rand_seed, int pretendRank, ior_dataPacketType_e dataPacketType){    
  if(dataPacketType == DATA_TIMESTAMP){
    cu_generate_memory_timestamp<<<bytes, 1>>>((float*)bytes, bytes, rand_seed, pretendRank);
  }
}
