#include "hip/hip_runtime.h"
/*
  This file contains CUDA code for creating and checking memory patterns on the device.
*/
#include <hip/hip_runtime.h>

#ifdef HAVE_CONFIG_H
#include "config.h"
#endif

#include <stdint.h>

#include "iordef.h"

#define RANDALGO_GOLDEN_RATIO_PRIME        0x9e37fffffffc0001UL

__global__ 
void cu_generate_memory_timestamp(uint64_t * buf, size_t length, int rand_seed, uint64_t pretendRank){
  size_t pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos < length){
    buf[pos] = pretendRank | rand_seed + pos;
  }
}

__global__ 
void cu_verify_memory_timestamp(uint64_t item, uint64_t * buf, size_t length, int rand_seed, uint64_t pretendRank, int * errors){
  
}

extern "C" void generate_memory_pattern_gpu(char * buf, size_t bytes, int rand_seed, int pretendRank, ior_dataPacketType_e dataPacketType){    
  size_t blocks = (bytes+2047)/2048;
  size_t threads = 256;
  if(dataPacketType == DATA_TIMESTAMP){    
    cu_generate_memory_timestamp<<<blocks, threads>>>((uint64_t*) buf, bytes/sizeof(uint64_t), rand_seed, ((uint64_t) pretendRank) << 32);
  }
}

extern "C" void update_write_memory_pattern_gpu(uint64_t item, char * buf, size_t bytes, int rand_seed, int rank, ior_dataPacketType_e dataPacketType){
  // nothing to do for dataPacketType == DATA_TIMESTAMP, i.e., won't be called for this parameter
  size_t blocks = (bytes+2047)/2048;
  size_t threads = 256;
}

extern "C" int verify_memory_pattern_gpu(uint64_t item, char * buffer, size_t bytes, int rand_seed, int pretendRank, ior_dataPacketType_e dataPacketType){
  int errors = 0;
  size_t blocks = (bytes+2047)/2048;
  size_t threads = 256;  
  if(dataPacketType == DATA_TIMESTAMP){
    cu_verify_memory_timestamp<<<blocks, threads>>>(item, (uint64_t*) buffer, bytes/sizeof(uint64_t), rand_seed, pretendRank, & errors);
  }
  return errors;
}
